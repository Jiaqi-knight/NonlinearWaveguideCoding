#include "hip/hip_runtime.h"
/* This code accompanies
 *   The Lattice Boltzmann Method: Principles and Practice
 *   T. Krüger, H. Kusumaatmaja, A. Kuzmin, O. Shardt, G. Silva, E.M. Viggen
 *   ISBN 978-3-319-44649-3 (Electronic) 
 *        978-3-319-44647-9 (Print)
 *   http://www.springer.com/978-3-319-44647-9
 *
 * This code is provided under the MIT license. See LICENSE.txt.
 *
 * Author: Orest Shardt
 *
 */
#include <stdio.h>
#define _USE_MATH_DEFINES
#include <math.h>

#include <hip/hip_runtime.h>

#include "LBM.h"

const int nThreads = 32;

__device__ __forceinline__ size_t gpu_field0_index(unsigned int x, unsigned int y)
{
    return NX*y+x;
}

__device__ __forceinline__ size_t gpu_scalar_index(unsigned int x, unsigned int y)
{
    return NX*y+x;
}

__device__ __forceinline__ size_t gpu_fieldn_index(unsigned int x, unsigned int y, unsigned int d)
{
    return (NX*(NY*(d-1)+y)+x);
}

#define checkCudaErrors(err)  __checkCudaErrors(err,#err,__FILE__,__LINE__)
#define getLastCudaError(msg)  __getLastCudaError(msg,__FILE__,__LINE__)

inline void __checkCudaErrors(hipError_t err, const char *const func, const char *const file, const int line )
{
    if(err != hipSuccess)
    {
        fprintf(stderr, "CUDA error at %s(%d)\"%s\": [%d] %s.\n",
                file, line, func, (int)err, hipGetErrorString(err));
        exit(-1);
    }
}

inline void __getLastCudaError(const char *const errorMessage, const char *const file, const int line )
{
    hipError_t err = hipGetLastError();
    if(err != hipSuccess) {
        fprintf(stderr, "CUDA error at %s(%d): [%d] %s.\n",
                file, line, (int)err, hipGetErrorString(err));
        exit(-1);
    }
}

// forward declarations of kernels
__global__ void gpu_taylor_green(unsigned int,double*,double*,double*);
__global__ void gpu_init_equilibrium(double*,double*,double*,double*,double*);
__global__ void gpu_stream_collide_save(double*,double*,double*,double*,double*,double*,bool);
__global__ void gpu_compute_flow_properties(unsigned int,double*,double*,double*,double*);

__device__ void taylor_green_eval(unsigned int t, unsigned int x, unsigned int y, double *r, double *u, double *v)
{
    double kx = 2.0*M_PI/NX;
    double ky = 2.0*M_PI/NY;
    double td = 1.0/(nu*(kx*kx+ky*ky));
    
    double X = x+0.5;
    double Y = y+0.5;
    double ux = -u_max*sqrt(ky/kx)*cos(kx*X)*sin(ky*Y)*exp(-1.0*t/td);
    double uy =  u_max*sqrt(kx/ky)*sin(kx*X)*cos(ky*Y)*exp(-1.0*t/td);
    double P = -0.25*rho0*u_max*u_max*((ky/kx)*cos(2.0*kx*X)+(kx/ky)*cos(2.0*ky*Y))*exp(-2.0*t/td);
    double rho = rho0+3.0*P;
    
    *r = rho;
    *u = ux;
    *v = uy;
}

__host__ void taylor_green(unsigned int t, double *r, double *u, double *v)
{
    // blocks in grid
    dim3  grid(NX/nThreads, NY, 1);
    // threads in block
    dim3  threads(nThreads, 1, 1);

    gpu_taylor_green<<< grid, threads >>>(t,r,u,v);
    getLastCudaError("gpu_taylor_green kernel error");
}

__global__ void gpu_taylor_green(unsigned int t, double *r, double *u, double *v)
{
    unsigned int y = blockIdx.y;
    unsigned int x = blockIdx.x*blockDim.x+threadIdx.x;
    
    size_t sidx = gpu_scalar_index(x,y);
    
    taylor_green_eval(t,x,y,&r[sidx],&u[sidx],&v[sidx]);
}

__host__ void init_equilibrium(double *f0, double *f1, double *r, double *u, double *v)
{
    // blocks in grid
    dim3  grid(NX/nThreads, NY, 1);
    // threads in block
    dim3  threads(nThreads, 1, 1);

    gpu_init_equilibrium<<< grid, threads >>>(f0,f1,r,u,v);
    getLastCudaError("gpu_init_equilibrium kernel error");
}

__global__ void gpu_init_equilibrium(double *f0, double *f1, double *r, double *u, double *v)
{
    unsigned int y = blockIdx.y;
    unsigned int x = blockIdx.x*blockDim.x+threadIdx.x;
    
    double rho = r[gpu_scalar_index(x,y)];
    double ux  = u[gpu_scalar_index(x,y)];
    double uy  = v[gpu_scalar_index(x,y)];
    
    // load equilibrium
    // feq_i  = w_i rho [1 + 3(ci . u) + (9/2) (ci . u)^2 - (3/2) (u.u)]
    // feq_i  = w_i rho [1 - 3/2 (u.u) + (ci . 3u) + (1/2) (ci . 3u)^2]
    // feq_i  = w_i rho [1 - 3/2 (u.u) + (ci . 3u){ 1 + (1/2) (ci . 3u) }]
    
    // temporary variables
    double w0r = w0*rho;
    double wsr = ws*rho;
    double wdr = wd*rho;
    double omusq = 1.0 - 1.5*(ux*ux+uy*uy);
    
    double tux = 3.0*ux;
    double tuy = 3.0*uy;
    
    f0[gpu_field0_index(x,y)]    = w0r*(omusq);
    
    double cidot3u = tux;
    f1[gpu_fieldn_index(x,y,1)]  = wsr*(omusq + cidot3u*(1.0+0.5*cidot3u));
    cidot3u = tuy;
    f1[gpu_fieldn_index(x,y,2)]  = wsr*(omusq + cidot3u*(1.0+0.5*cidot3u));
    cidot3u = -tux;
    f1[gpu_fieldn_index(x,y,3)]  = wsr*(omusq + cidot3u*(1.0+0.5*cidot3u));
    cidot3u = -tuy;
    f1[gpu_fieldn_index(x,y,4)]  = wsr*(omusq + cidot3u*(1.0+0.5*cidot3u));
    
    cidot3u = tux+tuy;
    f1[gpu_fieldn_index(x,y,5)]  = wdr*(omusq + cidot3u*(1.0+0.5*cidot3u));
    cidot3u = tuy-tux;
    f1[gpu_fieldn_index(x,y,6)]  = wdr*(omusq + cidot3u*(1.0+0.5*cidot3u));
    cidot3u = -(tux+tuy);
    f1[gpu_fieldn_index(x,y,7)]  = wdr*(omusq + cidot3u*(1.0+0.5*cidot3u));
    cidot3u = tux-tuy;
    f1[gpu_fieldn_index(x,y,8)]  = wdr*(omusq + cidot3u*(1.0+0.5*cidot3u));
}

__host__ void stream_collide_save(double *f0, double *f1, double *f2, double *r, double *u, double *v, bool save)
{
    // blocks in grid
    dim3  grid(NX/nThreads, NY, 1);
    // threads in block
    dim3  threads(nThreads, 1, 1);

    gpu_stream_collide_save<<< grid, threads >>>(f0,f1,f2,r,u,v,save);
    getLastCudaError("gpu_stream_collide_save kernel error");
}

__global__ void gpu_stream_collide_save(double *f0, double *f1, double *f2, double *r, double *u, double *v, bool save)
{
    // useful constants
    const double tauinv = 2.0/(6.0*nu+1.0); // 1/tau
    const double omtauinv = 1.0-tauinv;     // 1 - 1/tau

    unsigned int y = blockIdx.y;
    unsigned int x = blockIdx.x*blockDim.x+threadIdx.x;
    
    unsigned int xp1 = (x+1)%NX;
    unsigned int yp1 = (y+1)%NY;
    unsigned int xm1 = (NX+x-1)%NX;
    unsigned int ym1 = (NY+y-1)%NY;
    
    // direction numbering scheme
    // 6 2 5
    // 3 0 1
    // 7 4 8
    
    double ft0 = f0[gpu_field0_index(x,y)];
    
    // load populations from adjacent nodes
    double ft1 = f1[gpu_fieldn_index(xm1,y,  1)];
    double ft2 = f1[gpu_fieldn_index(x,  ym1,2)];
    double ft3 = f1[gpu_fieldn_index(xp1,y,  3)];
    double ft4 = f1[gpu_fieldn_index(x,  yp1,4)];
    double ft5 = f1[gpu_fieldn_index(xm1,ym1,5)];
    double ft6 = f1[gpu_fieldn_index(xp1,ym1,6)];
    double ft7 = f1[gpu_fieldn_index(xp1,yp1,7)];
    double ft8 = f1[gpu_fieldn_index(xm1,yp1,8)];
    
    // compute moments
    double rho = ft0+ft1+ft2+ft3+ft4+ft5+ft6+ft7+ft8;
    double rhoinv = 1.0/rho;
    
    double ux = rhoinv*(ft1+ft5+ft8-(ft3+ft6+ft7));
    double uy = rhoinv*(ft2+ft5+ft6-(ft4+ft7+ft8));
    
    // only write to memory when needed
    if(save)
    {
        r[gpu_scalar_index(x,y)] = rho;
        u[gpu_scalar_index(x,y)] = ux;
        v[gpu_scalar_index(x,y)] = uy;
    }
    
    // now compute and relax to equilibrium
    // note that
    // relax to equilibrium
    // feq_i  = w_i rho [1 + 3(ci . u) + (9/2) (ci . u)^2 - (3/2) (u.u)]
    // feq_i  = w_i rho [1 - 3/2 (u.u) + (ci . 3u) + (1/2) (ci . 3u)^2]
    // feq_i  = w_i rho [1 - 3/2 (u.u) + (ci . 3u){ 1 + (1/2) (ci . 3u) }]
    
    // temporary variables
    double tw0r = tauinv*w0*rho; //   w[0]*rho/tau 
    double twsr = tauinv*ws*rho; // w[1-4]*rho/tau
    double twdr = tauinv*wd*rho; // w[5-8]*rho/tau
    double omusq = 1.0 - 1.5*(ux*ux+uy*uy); // 1-(3/2)u.u
    
    double tux = 3.0*ux;
    double tuy = 3.0*uy;
    
    f0[gpu_field0_index(x,y)]    = omtauinv*ft0  + tw0r*(omusq);
    
    double cidot3u = tux;
    f2[gpu_fieldn_index(x,y,1)]  = omtauinv*ft1  + twsr*(omusq + cidot3u*(1.0+0.5*cidot3u));
    cidot3u = tuy;
    f2[gpu_fieldn_index(x,y,2)]  = omtauinv*ft2  + twsr*(omusq + cidot3u*(1.0+0.5*cidot3u));
    cidot3u = -tux;
    f2[gpu_fieldn_index(x,y,3)]  = omtauinv*ft3  + twsr*(omusq + cidot3u*(1.0+0.5*cidot3u));
    cidot3u = -tuy;
    f2[gpu_fieldn_index(x,y,4)]  = omtauinv*ft4  + twsr*(omusq + cidot3u*(1.0+0.5*cidot3u));
    
    cidot3u = tux+tuy;
    f2[gpu_fieldn_index(x,y,5)]  = omtauinv*ft5  + twdr*(omusq + cidot3u*(1.0+0.5*cidot3u));
    cidot3u = tuy-tux;
    f2[gpu_fieldn_index(x,y,6)]  = omtauinv*ft6  + twdr*(omusq + cidot3u*(1.0+0.5*cidot3u));
    cidot3u = -(tux+tuy);
    f2[gpu_fieldn_index(x,y,7)]  = omtauinv*ft7  + twdr*(omusq + cidot3u*(1.0+0.5*cidot3u));
    cidot3u = tux-tuy;
    f2[gpu_fieldn_index(x,y,8)]  = omtauinv*ft8  + twdr*(omusq + cidot3u*(1.0+0.5*cidot3u));
}

__host__ void compute_flow_properties(unsigned int t, double *r, double *u, double *v,
                             double *prop, double *prop_gpu, double *prop_host)
{
    // prop must point to space for 4 doubles:
    // 0: energy
    // 1: L2 error in rho
    // 2: L2 error in ux
    // 3: L2 error in uy
    
    // blocks in grid
    dim3  grid(NX/nThreads, NY, 1);
    // threads in block
    dim3  threads(nThreads, 1, 1);

    gpu_compute_flow_properties<<< grid, threads, 7*threads.x*sizeof(double) >>>(t,r,u,v,prop_gpu);
    getLastCudaError("gpu_compute_flow_properties kernel error");
    
    // transfer block sums to host memory
    size_t prop_size_bytes = 7*grid.x*grid.y*sizeof(double);
    checkCudaErrors(hipMemcpy(prop_host,prop_gpu,prop_size_bytes,hipMemcpyDeviceToHost));
    
    // initialise sums
    double E = 0.0; // kinetic energy
    
    double sumrhoe2 = 0.0; // sum of error squared in rho
    double sumuxe2 = 0.0;  //                         ux
    double sumuye2 = 0.0;  //                         uy
    
    double sumrhoa2 = 0.0; // sum of analytical rho squared
    double sumuxa2 = 0.0;  //                   ux
    double sumuya2 = 0.0;  //                   uy
    
    // finish summation with CPU
    for(unsigned int i = 0; i < grid.x*grid.y; ++i)
    {
        E += prop_host[7*i];
        sumrhoe2 += prop_host[7*i+1];
        sumuxe2  += prop_host[7*i+2];
        sumuye2  += prop_host[7*i+3];

        sumrhoa2 += prop_host[7*i+4];
        sumuxa2  += prop_host[7*i+5];
        sumuya2  += prop_host[7*i+6];
    }
    
    // compute and return final values
    prop[0] = E;
    prop[1] = sqrt(sumrhoe2/sumrhoa2);
    prop[2] = sqrt(sumuxe2/sumuxa2);
    prop[3] = sqrt(sumuye2/sumuya2);
}

__global__ void gpu_compute_flow_properties(unsigned int t, double *r, double *u, double *v, double *prop_gpu)
{
    unsigned int y = blockIdx.y;
    unsigned int x = blockIdx.x*blockDim.x+threadIdx.x;
    
    extern __shared__ double data[];
    
    // set up arrays for each variable
    // each array begins after the previous ends
    double *E     = data;
    double *rhoe2 = data +   blockDim.x;
    double *uxe2  = data + 2*blockDim.x;
    double *uye2  = data + 3*blockDim.x;
    double *rhoa2 = data + 4*blockDim.x;
    double *uxa2  = data + 5*blockDim.x;
    double *uya2  = data + 6*blockDim.x;
    
    // load density and velocity
    double rho = r[gpu_scalar_index(x,y)];
    double ux  = u[gpu_scalar_index(x,y)];
    double uy  = v[gpu_scalar_index(x,y)];
    
    // compute kinetic energy density
    E[threadIdx.x] = rho*(ux*ux + uy*uy);
    
    // compute analytical results
    double rhoa, uxa, uya;
    taylor_green_eval(t,x,y,&rhoa,&uxa,&uya);
    
    // compute terms for L2 error
    rhoe2[threadIdx.x] = (rho-rhoa)*(rho-rhoa);
    uxe2[threadIdx.x]  = (ux-uxa)*(ux-uxa);
    uye2[threadIdx.x]  = (uy-uya)*(uy-uya);
    
    rhoa2[threadIdx.x] = (rhoa-rho0)*(rhoa-rho0);
    uxa2[threadIdx.x]  = uxa*uxa;
    uya2[threadIdx.x]  = uya*uya;
    
    // synchronise data in shared memory
    __syncthreads();
    
    // only one thread proceeds
    if(threadIdx.x == 0)
    {
        // compute linear index for this block within grid
        size_t idx = 7*(gridDim.x*blockIdx.y+blockIdx.x);
        
        for(int n = 0; n  < 7; ++n)
            prop_gpu[idx+n] = 0.0;
        
        // sum values for this block from shared memory
        for(int i = 0; i < blockDim.x; ++i)
        {
            prop_gpu[idx  ] += E[i];
            prop_gpu[idx+1] += rhoe2[i];
            prop_gpu[idx+2] += uxe2[i];
            prop_gpu[idx+3] += uye2[i];

            prop_gpu[idx+4] += rhoa2[i];
            prop_gpu[idx+5] += uxa2[i];
            prop_gpu[idx+6] += uya2[i];
        }
    }
}

__host__ void report_flow_properties(unsigned int t, double *rho, double *ux, double *uy,
                                     double *prop_gpu, double *prop_host)
{
    double prop[4];
    compute_flow_properties(t,rho,ux,uy,prop,prop_gpu,prop_host);
    printf("%u,%g,%g,%g,%g\n",t,prop[0],prop[1],prop[2],prop[3]);
}

__host__ void save_scalar(const char* name, double *scalar_gpu, double *scalar_host, unsigned int n)
{
    // assume reasonably-sized file names
    char filename[128];
    char format[16];
    
    // compute maximum number of digits
    int ndigits = floor(log10((double)NSTEPS)+1.0);
    
    // generate format string
    // file name format is name0000nnn.bin
    sprintf(format,"%%s%%0%dd.bin",ndigits);
    sprintf(filename,format,name,n);
    
    // transfer memory from GPU to host
    checkCudaErrors(hipMemcpy(scalar_host,scalar_gpu,mem_size_scalar,hipMemcpyDeviceToHost));
    
    // open file for writing
    FILE *fout = fopen(filename,"wb+");
    
    // write data
    fwrite(scalar_host,1,mem_size_scalar,fout);
    
    // close file
    fclose(fout);
    
    if(ferror(fout))
    {
        fprintf(stderr,"Error saving to %s\n",filename);
        perror("");
    }
    else
    {
        if(!quiet)
            printf("Saved to %s\n",filename);
    }
}

