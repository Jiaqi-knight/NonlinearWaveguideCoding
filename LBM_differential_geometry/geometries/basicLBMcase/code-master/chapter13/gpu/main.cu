#include "hip/hip_runtime.h"
/* This code accompanies
 *   The Lattice Boltzmann Method: Principles and Practice
 *   T. Krüger, H. Kusumaatmaja, A. Kuzmin, O. Shardt, G. Silva, E.M. Viggen
 *   ISBN 978-3-319-44649-3 (Electronic) 
 *        978-3-319-44647-9 (Print)
 *   http://www.springer.com/978-3-319-44647-9
 *
 * This code is provided under the MIT license. See LICENSE.txt.
 *
 * Author: Orest Shardt
 *
 */
#include <stdio.h>
#include <stdlib.h>

#include "seconds.h"
#include "LBM.cu"

int main(int argc, char* argv[])
{
    printf("Simulating Taylor-Green vortex decay\n");
    printf("      domain size: %ux%u\n",NX,NY);
    printf("               nu: %g\n",nu);
    printf("              tau: %g\n",tau);
    printf("            u_max: %g\n",u_max);
    printf("             rho0: %g\n",rho0);
    printf("        timesteps: %u\n",NSTEPS);
    printf("       save every: %u\n",NSAVE);
    printf("    message every: %u\n",NMSG);
    printf("\n");
    
    double bytesPerMiB = 1024.0*1024.0;
    double bytesPerGiB = 1024.0*1024.0*1024.0;
    
    checkCudaErrors(hipSetDevice(0));
    int deviceId = 0;
    checkCudaErrors(hipGetDevice(&deviceId));
    
    hipDeviceProp_t deviceProp;
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, deviceId));
    
    size_t gpu_free_mem, gpu_total_mem;
    checkCudaErrors(hipMemGetInfo(&gpu_free_mem,&gpu_total_mem));
    
    printf("CUDA information\n");
    printf("       using device: %d\n", deviceId);
    printf("               name: %s\n",deviceProp.name);
    printf("    multiprocessors: %d\n",deviceProp.multiProcessorCount);
    printf(" compute capability: %d.%d\n",deviceProp.major,deviceProp.minor);
    printf("      global memory: %.1f MiB\n",deviceProp.totalGlobalMem/bytesPerMiB);
    printf("        free memory: %.1f MiB\n",gpu_free_mem/bytesPerMiB);
    printf("\n");
    
    double *f0_gpu,*f1_gpu,*f2_gpu;
    double *rho_gpu,*ux_gpu,*uy_gpu;
    double *prop_gpu;
    checkCudaErrors(hipMalloc((void**)&f0_gpu,mem_size_0dir));
    checkCudaErrors(hipMalloc((void**)&f1_gpu,mem_size_n0dir));
    checkCudaErrors(hipMalloc((void**)&f2_gpu,mem_size_n0dir));
    checkCudaErrors(hipMalloc((void**)&rho_gpu,mem_size_scalar));
    checkCudaErrors(hipMalloc((void**)&ux_gpu,mem_size_scalar));
    checkCudaErrors(hipMalloc((void**)&uy_gpu,mem_size_scalar));
    const size_t mem_size_props = 7*NX/nThreads*NY*sizeof(double);
    checkCudaErrors(hipMalloc((void**)&prop_gpu,mem_size_props));
    
    double *scalar_host  = (double*) malloc(mem_size_scalar);
    if(scalar_host == NULL)
    {
        fprintf(stderr,"Error: unable to allocate required host memory (%.1f MiB).\n",mem_size_scalar/bytesPerMiB);
        exit(-1);
    }
    
    size_t total_mem_bytes = mem_size_0dir + 2*mem_size_n0dir + 3*mem_size_scalar + mem_size_props;

    // create event objects
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    
    // compute Taylor-Green flow at t=0 
    // to initialise rho, ux, uy fields.
    taylor_green(0,rho_gpu,ux_gpu,uy_gpu);
    
    // initialise f1 as equilibrium for rho, ux, uy
    init_equilibrium(f0_gpu,f1_gpu,rho_gpu,ux_gpu,uy_gpu);
    
    save_scalar("rho",rho_gpu,scalar_host,0);
    save_scalar("ux", ux_gpu, scalar_host,0);
    save_scalar("uy", uy_gpu, scalar_host,0);
    
    if(computeFlowProperties)
    {
        report_flow_properties(0,rho_gpu,ux_gpu,uy_gpu,prop_gpu,scalar_host);
    }
    
    double begin = seconds();
    checkCudaErrors(hipEventRecord(start,0));
    
    // main simulation loop; take NSTEPS time steps
    for(unsigned int n = 0; n < NSTEPS; ++n)
    {
        bool save = (n+1)%NSAVE == 0;
        bool msg  = (n+1)%NMSG == 0;
        bool need_scalars = save || (msg && computeFlowProperties);
        
        // stream and collide from f1 storing to f2
        // optionally compute and save moments
        stream_collide_save(f0_gpu,f1_gpu,f2_gpu,rho_gpu,ux_gpu,uy_gpu,need_scalars);
        
        if(save)
        {
            save_scalar("rho",rho_gpu,scalar_host,n+1);
            save_scalar("ux", ux_gpu, scalar_host,n+1);
            save_scalar("uy", uy_gpu, scalar_host,n+1);
        }
        
        // swap pointers
        double *temp = f1_gpu;
        f1_gpu = f2_gpu;
        f2_gpu = temp;
        
        if(msg)
        {
            if(computeFlowProperties)
            {
                // note: scalar_host is big enough by a factor of nThreads/7
                report_flow_properties(n+1,rho_gpu,ux_gpu,uy_gpu,prop_gpu,scalar_host);
            }
            
            if(!quiet)
                printf("completed timestep %d\n",n+1);
        }
    }
    checkCudaErrors(hipEventRecord(stop,0));
    checkCudaErrors(hipEventSynchronize(stop));
    float milliseconds = 0.0f;
    checkCudaErrors(hipEventElapsedTime(&milliseconds,start,stop));
    
    double end = seconds();
    double runtime = end-begin;
    double gpu_runtime = 0.001*milliseconds;

    size_t doubles_read = ndir; // per node every time step
    size_t doubles_written = ndir;
    size_t doubles_saved = 3; // per node every NSAVE time steps
    
    // note NX*NY overflows when NX=NY=65536
    size_t nodes_updated = NSTEPS*size_t(NX*NY);
    size_t nodes_saved   = (NSTEPS/NSAVE)*size_t(NX*NY);
    double speed = nodes_updated/(1e6*runtime);
    
    double bandwidth = (nodes_updated*(doubles_read + doubles_written)+nodes_saved*(doubles_saved))*sizeof(double)/(runtime*bytesPerGiB);
    
    printf(" ----- performance information -----\n");
    printf("  memory allocated (GPU): %.1f (MiB)\n",total_mem_bytes/bytesPerMiB);
    printf(" memory allocated (host): %.1f (MiB)\n",mem_size_scalar/bytesPerMiB);
    printf("               timesteps: %u\n",NSTEPS);
    printf("           clock runtime: %.3f (s)\n",runtime);
    printf("             gpu runtime: %.3f (s)\n",gpu_runtime);
    printf("                   speed: %.2f (Mlups)\n",speed);
    printf("               bandwidth: %.1f (GiB/s)\n",bandwidth);
    
    // destory event objects
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));
    
    // free all memory allocatd on the GPU and host
    checkCudaErrors(hipFree(f0_gpu));
    checkCudaErrors(hipFree(f1_gpu));
    checkCudaErrors(hipFree(f2_gpu));
    checkCudaErrors(hipFree(rho_gpu));
    checkCudaErrors(hipFree(ux_gpu));
    checkCudaErrors(hipFree(uy_gpu));
    checkCudaErrors(hipFree(prop_gpu));    
    free(scalar_host);
    
    // release resources associated with the GPU device
    hipDeviceReset();
    
    return 0;
}

